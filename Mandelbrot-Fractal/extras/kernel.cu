#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <math.h>
#include <Windows.h>
#include <hip/hip_runtime.h>
#include <>
#include <GL/glew.h>
#include <GL/freeglut.h>
#include <cuda_gl_interop.h>

#pragma region Mandelbrot Fractal -> BMP Image

const int BYTES_PER_PIXEL = 3;   /// Red, Green, Blue
const int FILE_HEADER_SIZE = 14; /// Size in bytes
const int INFO_HEADER_SIZE = 40; /// Size in bytes

// Bitmap generation methods
void generateBitmapImage(unsigned char *image, int height, int width, char *file_name);
unsigned char *createBitmapFileHeader(int height, int stride);
unsigned char *createBitmapInfoHeader(int height, int width);

// Calculation of Z^2 + C
__device__ int iterateMandel(double c_real, double c_imag, int max_iters) {

	int n = 0;
	double real = c_real;
	double imag = c_imag;

	while (n < max_iters) {

		double real2 = real * real;
		double imag2 = imag * imag;
		imag = 2 * real * imag + c_imag;
		real = real2 - imag2 + c_real;

		if (real2 + imag2 > 4.0)
			return n;

		n++;
	}

	return -1;
}

// Main Fractal generation method (GPU)
__global__ void generateFractalGPU(int width, int height,
								   double x0, double x1,
								   double y0, double y1,
								   double pixel_width, double pixel_height,
								   int max_iters,
								   int *colors, size_t colors_amount,
								   unsigned char *image) {

	int py = blockIdx.y * blockDim.y + threadIdx.y;
	int px = blockIdx.x * blockDim.x + threadIdx.x;

	if ((px < width) && (py < height)) {

		double cY = y0 + py * pixel_height;
		double cX = x0 + px * pixel_width;

		int iter = iterateMandel(cX, cY, max_iters);

		int c = iter == -1 ? 0 : colors[iter % colors_amount];

		int index = py * width * BYTES_PER_PIXEL + (px * BYTES_PER_PIXEL);

		image[index + 2] = ((c >> 16) & 0xFF);  // Extract the RR byte
		image[index + 1] = ((c >> 8) & 0xFF);   // Extract the GG byte
		image[index] = ((c) & 0xFF);            // Extract the BB byte
	}
}

// Kernel wrapper
extern "C" {

	__host__ __declspec(dllexport) double generateFractalBMP(int width, int height,
														     double x0, double x1,
														     double y0, double y1,
														     double pixel_width, double pixel_height,
														     int max_iters,
														     int *colors, size_t colors_amount,
														     char *file_name) {

		// Timer variables
		LARGE_INTEGER frequency;
		LARGE_INTEGER begin, end;
		double gen_time, file_time;

		// Calculate full image (byte) size
		size_t img_size = width * height * BYTES_PER_PIXEL * sizeof(unsigned char);

		// Allocate memory with enough size to create the output image
		unsigned char *image = (unsigned char *)malloc(img_size);

		// Get ticks per second
		QueryPerformanceFrequency(&frequency);

		// Initialize device variables
		unsigned char *image_d;
		int *colors_d;

		// Get the byte size of available colors
		size_t colors_size = colors_amount * sizeof(int);

		// Allocate GPU memory
		hipMalloc(&image_d, img_size);
		hipMalloc(&colors_d, colors_size);

		// Copy data from host to device
		hipMemcpy(colors_d, colors, colors_size, hipMemcpyHostToDevice);

		// Define blocks and threads
		dim3 blocks(width / 32, height / 32);
		dim3 threads(32, 32);

		// Start timer
		QueryPerformanceCounter(&begin);

		// Execute the kernel
		generateFractalGPU <<<blocks, threads>>> (width, height, x0, x1, y0, y1, pixel_width, pixel_height, max_iters, colors_d, colors_amount, image_d);
		hipDeviceSynchronize();

		// Stop timer
		QueryPerformanceCounter(&end);

		// Copy data from device to host
		hipMemcpy(image, image_d, img_size, hipMemcpyDeviceToHost);

		// Free GPU memory
		hipFree(image_d);
		hipFree(colors_d);

		// Ellapsed time in milliseconds
		gen_time = (end.QuadPart - begin.QuadPart) * 1000.0 / frequency.QuadPart;
		printf("Fractal generated in %.3f ms\n", gen_time);

		// Start file timer
		QueryPerformanceCounter(&begin);

		// Generate the .bmp file
		generateBitmapImage(image, height, width, file_name);

		// Stop file timer
		QueryPerformanceCounter(&end);

		// Ellapsed time in milliseconds
		file_time = (end.QuadPart - begin.QuadPart) * 1000.0 / frequency.QuadPart;
		printf("BMP file generated in %.3f ms\n", file_time);

		// Free memory
		free(image);

		return gen_time;
	}
}

// Main BMP file generation method
void generateBitmapImage(unsigned char *image, int height, int width, char *file_name) {

	int width_in_bytes = width * BYTES_PER_PIXEL;

	unsigned char padding[3] = { 0, 0, 0 };
	int padding_size = (4 - (width_in_bytes) % 4) % 4;

	int stride = (width_in_bytes)+padding_size;

	FILE *image_file = fopen(file_name, "wb");

	unsigned char *file_header = createBitmapFileHeader(height, stride);
	fwrite(file_header, 1, FILE_HEADER_SIZE, image_file);

	unsigned char *info_header = createBitmapInfoHeader(height, width);
	fwrite(info_header, 1, INFO_HEADER_SIZE, image_file);

	// Write to file, line by line
	for (int i = 0; i < height; i++) {
		fwrite(image + (i * width_in_bytes), BYTES_PER_PIXEL, width, image_file);
		fwrite(padding, 1, padding_size, image_file);
	}

	fclose(image_file);
}

// Create the BMP file header
unsigned char *createBitmapFileHeader(int height, int stride) {

	int file_size = FILE_HEADER_SIZE + INFO_HEADER_SIZE + (stride * height);

	static unsigned char file_header[] = {
		0,0,     /// signature
		0,0,0,0, /// image file size in bytes
		0,0,0,0, /// reserved
		0,0,0,0, /// start of pixel array
	};

	file_header[0] = (unsigned char)('B');
	file_header[1] = (unsigned char)('M');
	file_header[2] = (unsigned char)(file_size);
	file_header[3] = (unsigned char)(file_size >> 8);
	file_header[4] = (unsigned char)(file_size >> 16);
	file_header[5] = (unsigned char)(file_size >> 24);
	file_header[10] = (unsigned char)(FILE_HEADER_SIZE + INFO_HEADER_SIZE);

	return file_header;
}

// Create the BMP file info header
unsigned char *createBitmapInfoHeader(int height, int width) {

	static unsigned char info_header[] = {
		0,0,0,0, /// header size
		0,0,0,0, /// image width
		0,0,0,0, /// image height
		0,0,     /// number of color planes
		0,0,     /// bits per pixel
		0,0,0,0, /// compression
		0,0,0,0, /// image size
		0,0,0,0, /// horizontal resolution
		0,0,0,0, /// vertical resolution
		0,0,0,0, /// colors in color table
		0,0,0,0, /// important color count
	};

	info_header[0] = (unsigned char)(INFO_HEADER_SIZE);
	info_header[4] = (unsigned char)(width);
	info_header[5] = (unsigned char)(width >> 8);
	info_header[6] = (unsigned char)(width >> 16);
	info_header[7] = (unsigned char)(width >> 24);
	info_header[8] = (unsigned char)(height);
	info_header[9] = (unsigned char)(height >> 8);
	info_header[10] = (unsigned char)(height >> 16);
	info_header[11] = (unsigned char)(height >> 24);
	info_header[12] = (unsigned char)(1);
	info_header[14] = (unsigned char)(BYTES_PER_PIXEL * 8);

	return info_header;
}

#pragma endregion

#pragma region Mandelbrot Fractal -> OpenGL

// The main window title
const char *main_title = "CUDA/OpenGL MandelBrot Fractal";

// The window/texture dimensions
int win_width = 800,
	win_height = 600;

// The window's aspect ratio
double aspect_ratio = 1.0;

// Fractal display variables
double2 center{ -0.75, 0.0 };
double scale = 1.0;
int iterations = 100;

// Cuda related variables
GLuint pbo = 0, tex = 0;
struct hipGraphicsResource *cuda_pbo_resource;

// Ultra fractal colors
__constant__ uchar3 colorMap[5] = {
	{0, 7, 100},
	{32, 107, 203},
	{237, 255, 255},
	{255, 170, 0},
	{0, 2, 0}
};

// Ultra fractal colors' points
__constant__ double colorPoints[5] = {
	0.0,
	0.16,
	0.42,
	0.6425,
	0.8575
};

// Transform window coordinates into fractal coordinates
__host__ __device__ double toFractalCoords(int n, int dimension, double scale, double extra = 1.0) {
	return (n - (dimension / 2.0)) * (scale / (dimension / 2.5) * extra);
}

// Returns a 0-1 double representing the distance in percentage between 2 colors
__device__ double calcParam(double t, double lp, double rp) {
	return (t - lp) / (rp - lp);
}

// Apply a hermite interpolation between 2 colors
__device__ uchar3 hermColor(int index, double param) {

	double t = param * param * (3.0f - 2.0f * param);

	return { (unsigned char)(((1.0 - t) * (double)colorMap[index].x) + (t * (double)colorMap[index + 1].x)),
			 (unsigned char)(((1.0 - t) * (double)colorMap[index].y) + (t * (double)colorMap[index + 1].y)),
			 (unsigned char)(((1.0 - t) * (double)colorMap[index].z) + (t * (double)colorMap[index + 1].z)) };
}

// Returns a color
__device__ uchar3 getColor(double t) {

	// Loop through color length - 1
	for (int i = 0; i < 4; i++) {

		// Verify if the given t is between 2 colors
		if (t >= colorPoints[i] && t < colorPoints[i + 1]) {

			// If so, calculate the percentage
			double param = calcParam(t, colorPoints[i], colorPoints[i + 1]);

			// And returned an interpolated color
			return hermColor(i, param);
		}
	}

	// If t is outside any 2 colors, return the last color
	return colorMap[4];
}

// Calculation of Z^2 + C (returns a color directly)
__device__ uchar3 iterateMandelOpenGL(double c_real, double c_imag, int max_iters) {

	int n = 0;
	double real = c_real;
	double imag = c_imag;

	while (n < max_iters) {

		double real2 = real * real;
		double imag2 = imag * imag;
		imag = 2.0 * real * imag + c_imag;
		real = real2 - imag2 + c_real;

		if (real2 + imag2 > 20.0) {
			// Apply color smoothing and return a color
			double sl = (double)(n + 4.0) - log2(log2(real2 + imag2)) / log2(2.0);
			return getColor(sl / (double)max_iters);
		}

		n++;
	}

	// Return black
	return {};
}

// Main Fractal generation method (GPU - OpenGL)
__global__ void generateFractalOpenGL(int width, int height,
									  double aspect_ratio,
									  double2 center,
									  double scale,
									  int max_iters,
									  uchar3 *fractal) {

	int px = blockIdx.x * blockDim.x + threadIdx.x;
	int py = blockIdx.y * blockDim.y + threadIdx.y;

	if ((px < width) && (py < height)) {

		// Invert Y because OpenGL's tex coordinates start at bottom left!
		double cX = toFractalCoords(px, width, scale, aspect_ratio) + center.x;
		double cY = -toFractalCoords(py, height, scale) + center.y;

		uchar3 color = iterateMandelOpenGL(cX, cY, max_iters);

		int index = py * width + px;

		fractal[index] = color;
	}
}

void initialize(int, char *[]);
void initWindow(int, char *[]);
void initPixelBuffer(void);
void render(void);
void drawTexture(void);
void resize(int, int);
void display(void);
void exitCudaInterop(void);
void mouseDrag(int, int);
void mousePress(int, int, int, int);
void mouseWheel(int, int, int, int);
void mouseDrag(int, int);

extern "C" {

	__host__ __declspec(dllexport) void mandelbrotFractalOpenGL(int width, int height) {

		// Create "empty" argc and argv to feed glutInit
		int argc = 1;
		char *argv[1] = { (char *)"Empty" };

		win_width = width;
		win_height = height;

		aspect_ratio = win_width / (double)win_height;

		initialize(argc, argv);

		glutMainLoop();

		atexit(exitCudaInterop);
	}
}

// Init OpenGL / Cuda
void initialize(int argc, char *argv[]) {

	initWindow(argc, argv);

	// Print OpenGL version used to the console
	fprintf(
		stdout,
		"INFO: OpenGL Version: %s\n",
		glGetString(GL_VERSION)
	);

	// Set up 2D orthographic region
	gluOrtho2D(0, win_width, win_height, 0);

	// Set up mouse functions
	glutMouseWheelFunc(mouseWheel);
	glutMouseFunc(mousePress);
	glutMotionFunc(mouseDrag);

	// Set up display functions
	glutReshapeFunc(resize);
	glutDisplayFunc(display);

	// Set up the cuda/opengl interop (pixel buffer / texture)
	initPixelBuffer();
}

// Don't forget to free the returned char* !!!
char *createWindowTitle() {

	// Create string buffer
	size_t buffer_size = 1024; // arbitrary buffer size
	char *buffer = (char *)malloc(buffer_size * sizeof(char));

	// Concatenate 'strings'
	snprintf(buffer, buffer_size, "%s | Resolution: %d x %d | Center: %.2f; %.2f | Iterations: %d | Scale: %.2f",
			 main_title, win_width, win_height, center.x, center.y, iterations, 1.0 / scale);

	// Return the buffer (pointer)
	return buffer;
}

// Setp up glut and glew
void initWindow(int argc, char *argv[]) {

	glutInit(&argc, argv);

	//glutInitContextVersion(3, 3);
	glutInitContextFlags(GLUT_FORWARD_COMPATIBLE);
	glutInitContextProfile(GLUT_CORE_PROFILE);

	glutSetOption(
		GLUT_ACTION_ON_WINDOW_CLOSE,
		GLUT_ACTION_GLUTMAINLOOP_RETURNS
	);

	glutInitWindowSize(win_width, win_height);

	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);

	char *window_title = createWindowTitle();

	int window_handle = glutCreateWindow(window_title);

	free(window_title);

	// Print error if the program couldn't create a window
	if (window_handle < 1) {
		fprintf(
			stderr,
			"ERROR: Could not create a new rendering window.\n"
		);
		getchar();
		exit(EXIT_FAILURE);
	}

	glewInit();
}

// Sets up the pixel buffer to be modified by CUDA and consumed by OpenGL
void initPixelBuffer() {

	glGenBuffers(1, &pbo);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	glBufferData(GL_PIXEL_UNPACK_BUFFER, BYTES_PER_PIXEL * win_width * win_height * sizeof(GLubyte), 0,
				 GL_STREAM_DRAW);
	glGenTextures(1, &tex);
	glBindTexture(GL_TEXTURE_2D, tex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo,
								 cudaGraphicsMapFlagsWriteDiscard);
}

// Handle window reshaping
void resize(int width, int height) {
	// Lock the display
	glutReshapeWindow(win_width, win_height);
}

// Handle OpenGL rendering
void render() {

	uchar3 *fractal_d = 0;

	hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
	hipGraphicsResourceGetMappedPointer((void **)&fractal_d, NULL, cuda_pbo_resource);

	// Define blocks and threads
	dim3 grid_size((win_width + 31) / 32, (win_height + 31) / 32);
	dim3 block_size(32, 32);

	// Execute the kernel
	generateFractalOpenGL <<<grid_size, block_size>>> (win_width, win_height, aspect_ratio, center, scale, iterations, fractal_d);
	hipDeviceSynchronize();

	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
}

// Draw a quad and texture with the same size as the display window
void drawTexture() {

	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, win_width, win_height, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
	glEnable(GL_TEXTURE_2D);
	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f); glVertex2f(0.0f, 0.0f);
	glTexCoord2f(0.0f, 1.0f); glVertex2f(0.0f, win_height);
	glTexCoord2f(1.0f, 1.0f); glVertex2f(win_width, win_height);
	glTexCoord2f(1.0f, 0.0f); glVertex2f(win_width, 0.0f);
	glEnd();
	glDisable(GL_TEXTURE_2D);
}

// The main display loop
void display() {

	glClear(GL_COLOR_BUFFER_BIT);
	render();
	drawTexture();
	glutSwapBuffers();
}

// Release CUDA resources/buffers
void exitCudaInterop() {

	if (pbo) {
		hipGraphicsUnregisterResource(cuda_pbo_resource);
		glDeleteBuffers(1, &pbo);
		glDeleteTextures(1, &tex);
	}
}

int pressed;
int2 startCoords;

// Handle mouse button pressing
void mousePress(int button, int state, int x, int y) {

	// Register the pressed button
	pressed = button;

	// Check if the mouse was pressed down
	if (state == GLUT_DOWN) {

		// Check left button press
		if (button == GLUT_LEFT_BUTTON) {

			// Save (window) coordinates where user clicked
			startCoords = { x, y };

		// Check right button press
		} else if (button == GLUT_RIGHT_BUTTON) {

			// Reset main variables
			center = { -0.75, 0.0 };
			scale = 1.0;
			iterations = 100;

			// Set new window title
			char *window_title = createWindowTitle();
			glutSetWindowTitle(window_title);
			free(window_title);
		}
	}
}

// Handle mouse dragging
void mouseDrag(int x, int y) {

	// Verify that user pressed left mouse button
	if (pressed == GLUT_LEFT_BUTTON) {

		// Move fractal center based on mouse movement
		int2 delta = { (x - startCoords.x) * aspect_ratio, (y - startCoords.y) * aspect_ratio };
		center.x -= delta.x * 0.002 * scale;
		center.y += delta.y * 0.002 * scale;
		startCoords = { x, y };

		// Set window title
		char *window_title = createWindowTitle();
		glutSetWindowTitle(window_title);
		free(window_title);

		// Call a redraw
		glutPostRedisplay();
	}
}

// Zoom in/out based on mouse position
void zoom(int dir, int x, int y) {

	// Save the old position (after converting window coordinates into fractal coordinates)
	double oldX = toFractalCoords(x, win_width, scale, aspect_ratio);
	double oldY = toFractalCoords(y, win_height, scale);

	// Apply scale change
	scale *= (1.0f - dir * 0.04);

	// Move center based on amount of zoom applied
	center.x -= toFractalCoords(x, win_width, scale, aspect_ratio) - oldX;
	center.y += toFractalCoords(y, win_height, scale) - oldY;
}

// Increase/Decrease ther number of iterations
void modifyIters(int dir) {
	iterations += dir * 10;
}

// Handle mouse wheel movement
void mouseWheel(int button, int dir, int x, int y) {

	// Get modifier key state for keyboard events
	int mod = glutGetModifiers();

	// Check if the user is pressing the SHIFT key
	if (mod == GLUT_ACTIVE_SHIFT) {
		// Change number of iterations if true
		modifyIters(dir);
	} else {
		// Apply zoom on fractal if false
		zoom(dir, x, y);
	}

	// Set new window title
	char *window_title = createWindowTitle();
	glutSetWindowTitle(window_title);
	free(window_title);

	// Call a redraw
	glutPostRedisplay();
}

#pragma endregion
